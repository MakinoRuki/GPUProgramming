#include "hip/hip_runtime.h"
#include "fft-cuda.cuh"

static __device__ __host__ inline float2 CplxAdd(float2 a, float2 b) {
    float2 c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

static __device__ __host__ inline float2 CplxMul(float2 a, float2 b) {
    float2 c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

static __device__ __host__ inline float2 CplxInv(float2 a) {
    float2 c;
    c.x = -a.x;
    c.y = -a.y;
    return c;
}

__global__ void bit_reorder(float2* da, float2* ra, int n, int s, int threads) {
    int id = blockIdx.x * threads + threadIdx.x;
    ra[id] = da[__brev(id) >> (32 - s)];
}

// __global__ void calc_fft(float2* ra, int k, int k_2, int start, int threads) {
//     int tid = blockIdx.x * threads + threadIdx.x;
//     int id = start * k_2 + tid;
//     int id2 = id + k;
//     float2 c;
//     c.x = __cosf((2.0 * M_PI * tid) / (1.0 * k_2));
//     c.y = -__sinf((2.0 * M_PI * tid) / (1.0 * k_2));
//     float2 u, t;
//     u = float2Add(ra[id], float2Mul(c, ra[id2]));
//     t = float2Add(ra[id], float2Inv(float2Mul(c, ra[id2])));
//     ra[id] = u;
//     ra[id2] = t;
// }

__global__ void calc_fft(float2* ra, int m, int threads) {
    int tid = blockIdx.x * threads + threadIdx.x;
    if (tid % m < m / 2) {
    	int id = tid;
	    int id2 = id + m / 2;
	    float2 c;
	    c.x = __cosf((2.0 * M_PI * (tid % m)) / (1.0 * m));
	    c.y = -__sinf((2.0 * M_PI * (tid % m)) / (1.0 * m));
	    float2 u, t;
	    u = CplxAdd(ra[id], CplxMul(c, ra[id2]));
	    t = CplxAdd(ra[id], CplxInv(CplxMul(c, ra[id2])));
	    ra[id] = u;
	    ra[id2] = t;
    }
}

void fft(float2* a, int n, int threads) {
    size_t data_size = n * sizeof(float2);
    float2* ra;
    float2* da;
    hipMalloc((void**)&ra, data_size);
    hipMalloc((void**)&da, data_size);
    hipMemcpy(da, a, data_size, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    int s = log2((float)n);
    bit_reorder<<<n/threads, threads>>>(da, ra, n, s, threads);

    // for (int i = 2; i < n; i *= 2) {
    //     for (int j = 0; j < n; j += i) {
    //         int k = i / 2;
    //         calc_fft<<<ceil(k/threads), threads>>>(ra, k, i, j, threads);
    //     }
    // }
    for (int i = 2; i < n; i *= 2) {
    	calc_fft<<<ceil(n/threads), threads>>>(ra, i, threads);
    }

    // float2* result;
    // result = (float2*)malloc(data_size);
    hipMemcpy(a, ra, data_size, hipMemcpyDeviceToHost);
    hipFree(da);
    hipFree(ra);
}
