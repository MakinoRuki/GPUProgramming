#include "hip/hip_runtime.h"
#include "fft-cuda.cuh"

static __device__ __host__ inline Cplx CplxAdd(Cplx& a, Cplx& b) {
    Cplx c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

static __device__ __host__ inline Cplx CplxMul(Cplx& a, Cplx& b) {
    Cplx c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

static __device__ __host__ inline Cplx CplxInv(Cplx& a) {
    Cplx c(-a.x, -a.y);
    return c;
}

__global__ void bit_reorder(Cplx* da, Cplx* ra, int n, int s, int threads) {
    int id = blockIdx.x * threads + threadIdx.x;
    ra[id] = da[__brev(id) >> (32 - s)];
}

// __global__ void calc_fft(Cplx* ra, int k, int k_2, int start, int threads) {
//     int tid = blockIdx.x * threads + threadIdx.x;
//     int id = start * k_2 + tid;
//     int id2 = id + k;
//     Cplx c;
//     c.x = __cosf((2.0 * M_PI * tid) / (1.0 * k_2));
//     c.y = -__sinf((2.0 * M_PI * tid) / (1.0 * k_2));
//     Cplx u, t;
//     u = CplxAdd(ra[id], CplxMul(c, ra[id2]));
//     t = CplxAdd(ra[id], CplxInv(CplxMul(c, ra[id2])));
//     ra[id] = u;
//     ra[id2] = t;
// }

__global__ void calc_fft(Cplx* ra, int m, int threads) {
    int tid = blockIdx.x * threads + threadIdx.x;
    if (tid % m < m / 2) {
    	int id = tid;
	    int id2 = id + m / 2;
	    Cplx c;
	    c.x = __cosf((2.0 * M_PI * (tid % m)) / (1.0 * m));
	    c.y = -__sinf((2.0 * M_PI * (tid % m)) / (1.0 * m));
	    Cplx u, t;
	    u = CplxAdd(ra[id], CplxMul(c, ra[id2]));
	    t = CplxAdd(ra[id], CplxInv(CplxMul(c, ra[id2])));
	    ra[id] = u;
	    ra[id2] = t;
    }
}

void fft(Cplx* a, int n, int threads) {
    size_t data_size = n * sizeof(Cplx);
    Cplx* ra, da;
    hipMalloc((void**)&ra, data_size);
    hipMalloc((void**)&da, data_size);
    hipMemcpy(da, a, data_size, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    int s = log2(n);
    bit_reorder<<<n/threads, threads>>>(da, ra, n, s, threads);

    // for (int i = 2; i < n; i *= 2) {
    //     for (int j = 0; j < n; j += i) {
    //         int k = i / 2;
    //         calc_fft<<<ceil(k/threads), threads>>>(ra, k, i, j, threads);
    //     }
    // }
    for (int i = 2; i < n; i *= 2) {
    	calc_fft<<<ceil(n/threads), threads>>>(ra, i, threads);
    }

    Cplx* result;
    result = (Cplx*)malloc(data_size);
    hipMemcpy(result, ra, data_size, hipMemcpyDeviceToHost);
    hipFree(da);
    hipFree(ra);
}
